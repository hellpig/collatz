#include "hip/hip_runtime.h"
/* ******************************************

Sieves of size 2^k are used, where k can be very large!

On Linux, compile via something like...
  nvcc collatzPartiallySieveless_repeatedKsteps_CUDA.cu

Note that I use the "long long" function strtoull when reading in the arguments.

Currently loads in a sieve file, which must must match the k1 value set in this code.

Currently requires my cuda_uint128.h

k < 81 must be true




Parts of the kernels in this file are modified from...
  https://github.com/xbarin02/collatz/blob/master/src/gpuworker/kernel32-precalc.cl

I am using the idea of this paper...
  http://www.ijnc.org/index.php/ijnc/article/download/135/144



This code requires requires two arguments...
  ./a.out  TASK_ID_KERNEL2  TASK_ID

Starting at 0, increase TASK_ID by 1 each run until its max
  ( 2^(k - TASK_SIZE) - 1 )
Then, reset TASK_ID and increase TASK_ID_KERNEL2 by 1 (starting at 0).
Don't skip!

Here is a neat way to run this code...
  seq -f %1.0f 0 9 | xargs -L 1 -P 1 ./a.out 0 |tee -a log.txt &
To pick up where you left off, change the start and stop value of seq.
k and TASK_SIZE_KERNEL2 should not change between runs.

For each TASK_ID_KERNEL2, 9 * 2 ^ TASK_SIZE_KERNEL2 numbers will be tested,
  but only after each TASK_ID is run from 0 to ( 2^(k - TASK_SIZE) - 1 )
Why the 9? I thought it might help my GPU code, but it only does EXTREMELY SLIGHTLY.
Feel free to get rid of the 9 when h_minimum and h_supremum are defined in
  kernel2
You'll also want to get rid of the division by 9 when this host program
  checks if TASK_ID_KERNEL2 will cause certain overflow.
If TASK_ID_KERNEL2 > 0, you'll also want to fix the line in kernel2...
  aMod = 0



I have assumed that your computer is 64-bit.
If not, this will limit various run parameters.

This code is not compatible with a watchdog timer.
I recommend that you play around with your device to see if it has a GPU watchdog timer.
It has one if the kernel always stops after a certain amount of time (like 5 seconds).

I would expect that, if a bunch of OVERFLOW messages are printed by kernel2_repeatedKsteps
  within a short period of time, some might be lost due to buffer sizes ??

274133054632352106267 will overflow 128 bits and, for not-crazy k2, should be
  noticed by this code!
This number is over 64-bits, so it cannot be made into an integer literal.
To test 128-bit overflow and using printf() in kernel2,
calculate the following using Python 3, then put it as your TASK_ID_KERNEL2...
  TASK_ID_KERNEL2 = 274133054632352106267 // (9 << TASK_SIZE_KERNEL2)
Then set the following as your TASK_ID...
  remainder = (274133054632352106267 % (9 << TASK_SIZE_KERNEL2))
  TASK_ID = (remainder % (1 << k)) // (1 << TASK_SIZE)


(c) 2021 Bradley Knockel

****************************************** */


#include "cuda_uint128.h"

#include <unistd.h>   // for usleep()

#include <sys/time.h>
struct timeval tv1, tv2;



/*
  For a 2^k sieve
  k < 81
*/

#define SIEVE_LOGSIZE 51
const int k = SIEVE_LOGSIZE;


/*
  For a 2^k2 sieve to do k2 steps at a time after the initial k steps
  3 < k2 < 37
  Will use more than 2^(k2 + 3) bytes of RAM
  For my GPU, 18 is the best because it fits in GPU cache
*/

#define SIEVE_LOGSIZE2 18
const int k2 = SIEVE_LOGSIZE2;



/*
  For kernel1 and kernel1_2, which make the sieves...
    TASK_UNITS + 8 <= TASK_SIZE <= k
    TASK_UNITS <= k2
  Will use more than 2^TASK_SIZE bytes of RAM
  Due to the limits of strtoull(), k - TASK_SIZE < 64
  2^TASK_UNITS should be much larger than the number of CUDA Cores you have
  (fraction of 2^k sieve not excluded) * 2^TASK_SIZE should be much larger than the number of CUDA Cores you have
*/

#define TASK_SIZE 24
#define TASK_UNITS 16



/*
  TASK_SIZE_KERNEL2 - k should ideally be at least 10
  9 * 2^TASK_SIZE_KERNEL2 numbers will be run
  For kernel2, which tests numbers given a sieve...
  9 * 2^(TASK_SIZE_KERNEL2 + TASK_SIZE - k) numbers will be run by each process
  9 * 2^TASK_SIZE_KERNEL2 numbers will be run total by all processes
*/

#define TASK_SIZE_KERNEL2 67






// For the 2^k1 sieve that speeds up the calculation of the 2^k sieve...
//   TASK_SIZE <= k1 <= k
// The only con of a larger k1 is that the sieve file is harder to create and store,
//   but, once you have the sieve file, use it!
// Especially good values are 32, 35, 37, 40, ...

const int k1 = 37;

const char file[10] = "sieve37";




// integer that is 2^(non-negative integer), so 1, 2, 4, 8, ...
// For setting the number of CUDA threads per block
// Currently, this controls all three kernels
// Note that 2^TASK_UNITS must be at least (multipleOf32 * 2^5), but this shouldn't be a problem
// Note that (2^TASK_UNITS / (multipleOf32 * 2^5) ) cannot exceed the max unsigned int
//   because blockIdx.x is an unsigned int (again, not a problem)

#define multipleOf32 1





/*
  just for kernels
*/

#define LUT_SIZE32 21

#define UINT128_MAX (~(uint128_t)0)

__device__ uint32_t pow3(size_t n)
{
	uint32_t r = 1;
	uint32_t b = 3;

	while (n) {
		if (n & 1) {
			r *= b;
		}
		b *= b;
		n >>= 1;
	}

	return r;
}

/* 2^8 sieve */
__constant__ uint8_t sieve8[16];
const uint8_t sieve8_host[16] = {27, 31, 47, 71, 91, 103, 111, 127, 155, 159, 167, 191, 231, 239, 251, 255};







/*
  kernel1 makes k1 sieve
*/


__global__ void kernel1(
	uint16_t *arraySmall,    // array length is 2^(TASK_SIZE - 8)
	uint64_t *arrayLarge,    // array length is 2^(TASK_SIZE - 3)
	uint8_t *arrayIncreases, // array length is 2^(TASK_SIZE - 4)
	uint64_t TASK_ID
)
{
	size_t id = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;


	__shared__ uint32_t lut[LUT_SIZE32];

	if (threadIdx.x == 0) {
		for (size_t i = 0; i < LUT_SIZE32; ++i) {
			lut[i] = pow3(i);
		}
	}

	__syncthreads();


	size_t i_minimum  = ((size_t)(id + 0) << (TASK_SIZE - TASK_UNITS - 4));
	size_t i_supremum = ((size_t)(id + 1) << (TASK_SIZE - TASK_UNITS - 4));


	/* loop over bits in arraySmall[] */
	size_t pattern = i_minimum >> 4;  /* a group of 256 numbers */
	int bit = 0;                      /* which of the 16 possible numbers in the 2^8 sieve */
	for (size_t index = i_minimum; index < i_supremum; ++index) {

		/* search 2^k1 sieve for next number that could require testing */
		while ( !( arraySmall[pattern] & ((int)1 << bit) ) ) {
			index++;
			if (bit < 15)
				bit++;
			else {
				bit = 0;
				pattern++;
			}
		}
		if (index >= i_supremum) {
			break;
		}

		uint128_t L0 = ((uint128_t)TASK_ID << TASK_SIZE) + (pattern * 256 + sieve8[bit]);

		uint128_t L = L0;
		uint32_t Salpha = 0;      /* sum of alpha */

		uint128_t nextL = L0 - 1;
		uint32_t nextSalpha = 0;

		int R = SIEVE_LOGSIZE;  /* counter */
		do {
			/* note that L starts as odd, so we start by increasing */
			++L;
			do {
				//uint32_t alpha = (size_t)ctz((uint32_t)L.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = (__ffs((uint32_t)L.lo) - 1);

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				alpha = min(alpha, (uint32_t)R);
				R -= alpha;
				L >>= alpha;
				L *= lut[alpha];
				Salpha += alpha;
				if (R == 0) {
					--L;
					goto next1;
				}
			} while (!(L.lo & 1));
			L.lo--;    // L is odd
			do {
				//size_t beta = ctz((uint32_t)L.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t beta = __ffs((uint32_t)L.lo) - 1;

				beta = min(beta, (uint32_t)R);
				R -= beta;
				L >>= beta;
				if (L < L0) goto next3;
				if (R == 0) goto next1;
			} while (!(L.lo & 1));
		} while (1);

next1:

		/* 
		  Try something else.
		  See if L0 - 1 joins paths in SIEVE_LOGSIZE steps
		  Is it worth doing this ???
		*/

		R = SIEVE_LOGSIZE;
		do {
			/* note that nextL starts as even, so we start by decreasing */
			do {
				//size_t beta = ctz((uint32_t)nextL.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t beta = __ffs((uint32_t)nextL.lo) - 1;

				beta = min(beta, (uint32_t)R);
				R -= beta;
				nextL >>= beta;
				if (R == 0) goto next2;
			} while (!(nextL.lo & 1));
			++nextL;
			do {
				//uint32_t alpha = (size_t)ctz((uint32_t)nextL.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = __ffs((uint32_t)nextL.lo) - 1;

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				alpha = min(alpha, (uint32_t)R);
				R -= alpha;
				nextL >>= alpha;
				nextL *= lut[alpha];
				nextSalpha += alpha;
				if (R == 0) {
					--nextL;
					goto next2;
				}
			} while (!(nextL.lo & 1));
			nextL.lo--;   // nextL is odd
		} while (1);


next2:

		/* only write to RAM if number needs to be tested */
		if ( L == nextL && Salpha == nextSalpha) goto next3;

		arrayLarge[2*index] = L.hi;
		arrayLarge[2*index + 1] = L.lo;
		arrayIncreases[index] = (uint8_t)Salpha;

		//printf(" %llu\n", L0.lo);

next3:

		if (bit < 15)
			bit++;
		else {
			bit = 0;
			pattern++;
		}

	}

}










/*
  kernel1_2 makes k2 sieve
*/


__global__ void kernel1_2(
	uint64_t *arrayLarge2     // array length is 2^SIEVE_LOGSIZE2
)
{
	size_t id = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;


	__shared__ uint32_t lut[LUT_SIZE32];

	if (threadIdx.x == 0) {
		for (size_t i = 0; i < LUT_SIZE32; ++i) {
			lut[i] = pow3(i);
		}
	}

	__syncthreads();


	size_t i_minimum  = ((size_t)(id + 0) << (SIEVE_LOGSIZE2 - TASK_UNITS));
	size_t i_supremum = ((size_t)(id + 1) << (SIEVE_LOGSIZE2 - TASK_UNITS));


	for (size_t index = i_minimum; index < i_supremum; ++index) {

		uint64_t L = index;   // index is the initial L

		int R = SIEVE_LOGSIZE2;  /* counter */

		uint32_t Salpha = 0; /* sum of alpha */

		if (L == 0) goto next;

		do {
			L++;
			do {
				//uint32_t alpha = (size_t)ctz((uint32_t)L);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = __ffs((uint32_t)L) - 1;

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				alpha = min(alpha, (uint32_t)R);
				R -= alpha;
				L >>= alpha;
				L *= lut[alpha];
				Salpha += alpha;
				if (R == 0) {
					L--;
					goto next;
				}
			} while (!(L & 1));
			L--;
			do {
				//size_t beta = ctz((uint32_t)L);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t beta = __ffs((uint32_t)L) - 1;

				beta = min(beta, (uint32_t)R);
				R -= beta;
				L >>= beta;
				if (R == 0) goto next;
			} while (!(L & 1));
		} while (1);

next:

		/* stores both L and Salpha */
		arrayLarge2[index] = L + ((uint64_t)Salpha << 58);

	}

}






/*
  kernel2 uses the sieves
*/

__global__ void kernel2(
	uint64_t *indices,
	uint64_t *arrayLarge,      // actually 128-bit integers
	uint8_t *arrayIncreases,
	uint64_t *arrayLarge2,
	uint64_t TASK_ID,
	uint64_t TASK_ID_KERNEL2
)
{
	size_t id = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;

	__shared__ uint64_t lut[SIEVE_LOGSIZE2 + 1];
	__shared__ uint128_t maxNs[SIEVE_LOGSIZE2 + 1];

	if (threadIdx.x == 0) {
		for (size_t i = 0; i < SIEVE_LOGSIZE2 + 1; ++i) {
			lut[i] = pow3(i);
			//maxNs[i] = UINT128_MAX / lut[i];
		}
		maxNs[0].hi = 18446744073709551615;
		maxNs[0].lo = 18446744073709551615;
		maxNs[1].hi = 6148914691236517205;
		maxNs[1].lo = 6148914691236517205;
		maxNs[2].hi = 2049638230412172401;
		maxNs[2].lo = 14347467612885206812;
		maxNs[3].hi = 683212743470724133;
		maxNs[3].lo = 17080318586768103348;
		maxNs[4].hi = 227737581156908044;
		maxNs[4].lo = 11842354220159218321;
#if (SIEVE_LOGSIZE2 >= 5)
		maxNs[5].hi = 75912527052302681;
		maxNs[5].lo = 10096366097956256645;
#endif
#if (SIEVE_LOGSIZE2 >= 6)
		maxNs[6].hi = 25304175684100893;
		maxNs[6].lo = 15663284748458453292;
#endif
#if (SIEVE_LOGSIZE2 >= 7)
		maxNs[7].hi = 8434725228033631;
		maxNs[7].lo = 5221094916152817764;
#endif
#if (SIEVE_LOGSIZE2 >= 8)
		maxNs[8].hi = 2811575076011210;
		maxNs[8].lo = 7889279663287456460;
#endif
#if (SIEVE_LOGSIZE2 >= 9)
		maxNs[9].hi = 937191692003736;
		maxNs[9].lo = 14927589270235519897;
#endif
#if (SIEVE_LOGSIZE2 >= 10)
		maxNs[10].hi = 312397230667912;
		maxNs[10].lo = 4975863090078506632;
#endif
#if (SIEVE_LOGSIZE2 >= 11)
		maxNs[11].hi = 104132410222637;
		maxNs[11].lo = 7807535721262686082;
#endif
#if (SIEVE_LOGSIZE2 >= 12)
		maxNs[12].hi = 34710803407545;
		maxNs[12].lo = 14900341289560596438;
#endif
#if (SIEVE_LOGSIZE2 >= 13)
		maxNs[13].hi = 11570267802515;
		maxNs[13].lo = 4966780429853532146;
#endif
#if (SIEVE_LOGSIZE2 >= 14)
		maxNs[14].hi = 3856755934171;
		maxNs[14].lo = 13953422859090878459;
#endif
#if (SIEVE_LOGSIZE2 >= 15)
		maxNs[15].hi = 1285585311390;
		maxNs[15].lo = 10800055644266810025;
#endif
#if (SIEVE_LOGSIZE2 >= 16)
		maxNs[16].hi = 428528437130;
		maxNs[16].lo = 3600018548088936675;
#endif
#if (SIEVE_LOGSIZE2 >= 17)
		maxNs[17].hi = 142842812376;
		maxNs[17].lo = 13497835565169346635;
#endif
#if (SIEVE_LOGSIZE2 >= 18)
		maxNs[18].hi = 47614270792;
		maxNs[18].lo = 4499278521723115545;
#endif
#if (SIEVE_LOGSIZE2 >= 19)
		maxNs[19].hi = 15871423597;
		maxNs[19].lo = 7648674198477555720;
#endif
#if (SIEVE_LOGSIZE2 >= 20)
		maxNs[20].hi = 5290474532;
		maxNs[20].lo = 8698472757395702445;
#endif
#if (SIEVE_LOGSIZE2 >= 21)
		maxNs[21].hi = 1763491510;
		maxNs[21].lo = 15197320301604935225;
#endif
#if (SIEVE_LOGSIZE2 >= 22)
		maxNs[22].hi = 587830503;
		maxNs[22].lo = 11214688125104828947;
#endif
#if (SIEVE_LOGSIZE2 >= 23)
		maxNs[23].hi = 195943501;
		maxNs[23].lo = 3738229375034942982;
#endif
#if (SIEVE_LOGSIZE2 >= 24)
		maxNs[24].hi = 65314500;
		maxNs[24].lo = 7394991149581498199;
#endif
#if (SIEVE_LOGSIZE2 >= 25)
		maxNs[25].hi = 21771500;
		maxNs[25].lo = 2464997049860499399;
#endif
#if (SIEVE_LOGSIZE2 >= 26)
		maxNs[26].hi = 7257166;
		maxNs[26].lo = 13119495065759867543;
#endif
#if (SIEVE_LOGSIZE2 >= 27)
		maxNs[27].hi = 2419055;
		maxNs[27].lo = 10522079713156473053;
#endif
#if (SIEVE_LOGSIZE2 >= 28)
		maxNs[28].hi = 806351;
		maxNs[28].lo = 15805189286858525428;
#endif
#if (SIEVE_LOGSIZE2 >= 29)
		maxNs[29].hi = 268783;
		maxNs[29].lo = 17566225811425876220;
#endif
#if (SIEVE_LOGSIZE2 >= 30)
		maxNs[30].hi = 89594;
		maxNs[30].lo = 12004323295045142612;
#endif
#if (SIEVE_LOGSIZE2 >= 31)
		maxNs[31].hi = 29864;
		maxNs[31].lo = 16299270480821415281;
#endif
#if (SIEVE_LOGSIZE2 >= 32)
		maxNs[32].hi = 9954;
		maxNs[32].lo = 17730919542746839504;
#endif
#if (SIEVE_LOGSIZE2 >= 33)
		maxNs[33].hi = 3318;
		maxNs[33].lo = 5910306514248946501;
#endif
#if (SIEVE_LOGSIZE2 >= 34)
		maxNs[34].hi = 1106;
		maxNs[34].lo = 1970102171416315500;
#endif
#if (SIEVE_LOGSIZE2 >= 35)
		maxNs[35].hi = 368;
		maxNs[35].lo = 12954530106278472910;
#endif
#if (SIEVE_LOGSIZE2 >= 36)
		maxNs[36].hi = 122;
		maxNs[36].lo = 16616006084565858714;
#endif
	}

	__syncthreads();

	uint128_t h_minimum = ((uint128_t)(TASK_ID_KERNEL2 + 0)*9 << (TASK_SIZE_KERNEL2 - SIEVE_LOGSIZE));
	uint128_t h_supremum = ((uint128_t)(TASK_ID_KERNEL2 + 1)*9 << (TASK_SIZE_KERNEL2 - SIEVE_LOGSIZE));

	//int cMod = ((uint128_t)1 << SIEVE_LOGSIZE) % 3;
	int cMod = (SIEVE_LOGSIZE & 1) ? 2 : 1 ;



	size_t index = indices[id];
	if (index == (~(uint64_t)0)) return;

	// deal with the 2^8 sieve that is used to compress the 2^k1 sieve
        //        L0 = TASK_ID * 2^TASK_SIZE             + (index / 16) * 256   + sieve8[index % 16]
	uint128_t L0 = ((uint128_t)TASK_ID << TASK_SIZE) + (((index >> 4) << 8) + sieve8[index & 0xf]);

	int aMod = 0;
	//int bMod = L0 % 3;

        /* trick for mod 3 */
	uint64_t r = 0;
	r += (uint32_t)(L0.lo);
	r += (L0.lo >> 32);
	r += (uint32_t)(L0.hi);
	r += (L0.hi >> 32);
        int bMod = r%3;



	/* precalculate */
	uint128_t L;
	L.hi = arrayLarge[2*index];
        L.lo = arrayLarge[2*index + 1];
	uint32_t Salpha = (uint32_t)arrayIncreases[index];



	/* iterate over highest bits */
	for (uint128_t h = h_minimum; h < h_supremum; ++h) {
		uint128_t N0 = (h << SIEVE_LOGSIZE) + L0;
		int N0Mod = (aMod * cMod + bMod) % 3;
		aMod++;
		if (aMod > 2) aMod -= 3;

		while (N0Mod == 2) {
			++h;
			N0 = (h << SIEVE_LOGSIZE) + L0;
			N0Mod = (aMod * cMod + bMod) % 3;
			aMod++;
			if (aMod > 2) aMod -= 3;
		}
		if (h >= h_supremum) {
			break;
		}



		/* find the N after the first k steps */
		uint128_t N = h;
		uint32_t Salpha0 = Salpha;
		do {
			uint32_t alpha = min(Salpha0, (uint32_t)SIEVE_LOGSIZE2);
			N *= lut[alpha];
			Salpha0 -= alpha;
		} while (Salpha0 > 0);
		N += L;


		do {
			index = N.lo & ( ((uint64_t)1<<SIEVE_LOGSIZE2) - 1 );  // get lowest k2 bits of N
			uint64_t newL = arrayLarge2[index];
			size_t newSalpha = newL >> 58;    // just 6 bits gives Salpha
			newL &= 0x3ffffffffffffff;        // rest of bits gives L

			/* find the new N */
			N >>= SIEVE_LOGSIZE2;
			if (N > maxNs[newSalpha]) {
				printf("  OVERFLOW: (%llu << 64) | %llu\n", N0.hi, N0.lo);
				break;
			}
			N *= lut[newSalpha];
			if (N > UINT128_MAX - newL) {
				printf("  OVERFLOW: (%llu << 64) | %llu\n", N0.hi, N0.lo);
				break;
			}
			N += newL;

			if (N < N0) break;
		} while (1);

	}


}







int main(int argc, char *argv[]) {


  if( argc < 3 ) {
    printf("Too few arguments. Aborting.\n");
    return 0;
  }

  uint64_t TASK_ID_KERNEL2 = (uint64_t)strtoull(argv[1], NULL, 10);

  uint64_t TASK_ID         = (uint64_t)strtoull(argv[2], NULL, 10);

  uint64_t maxTaskID = ((uint64_t)1 << (k - TASK_SIZE));
  if ( TASK_ID >= maxTaskID ) {
    printf("Aborting. TASK_ID must be less than %" PRIu64 "\n", maxTaskID);
    return 0;
  }


  /* Check for 100%-certain overflow.
     This check prevents having to check when doing any pre-calculation when interlacing
     Note that after k steps, for A * 2^k + B...
       B = 2^k - 1 will become 3^k - 1
       and A*2^k will become A*3^k
  */
  __uint128_t temp = 1;  // will equal 3^k
  __uint128_t UINT128_MAX2 = (~(__uint128_t)0);
  for (int i=0; i < k; i++) { temp *= 3; }
  if ( (__uint128_t)TASK_ID_KERNEL2 > ((UINT128_MAX2 - temp) >> (TASK_SIZE_KERNEL2 - k)) / 9 / temp - 1 ) {
    printf("  Well, aren't you ambitious!\n");
    return 0;
  }




  printf("TASK_ID_KERNEL2 = %" PRIu64 "\n", TASK_ID_KERNEL2);
  printf("TASK_ID = %" PRIu64 "\n", TASK_ID);
  printf("TASK_ID must be less than %" PRIu64 "\n", maxTaskID);
  printf("TASK_SIZE_KERNEL2 = %i\n", TASK_SIZE_KERNEL2);
  printf("TASK_SIZE = %i\n", TASK_SIZE);
  printf("  k = %i\n", k);
  printf("  k1 = %i\n", k1);
  printf("  k2 = %i\n", k2);
  fflush(stdout);




  // start timing
  gettimeofday(&tv1, NULL);



	/* setup kernel stuff */

	hipError_t ret;

	size_t arraySmallCount = ((size_t)1 << (TASK_SIZE - 8)) + 1;   // each element is 2^8 numbers; add 1 to prevent kern1 from reading too far
	size_t arrayLargeCount = (size_t)1 << (TASK_SIZE - 3);     // two of these are needed per uint128
	size_t arrayLarge2Count = (size_t)1 << k2;                 // for k2
	size_t arrayIncreasesCount = (size_t)1 << (TASK_SIZE - 4); // one for each bit in portion of 2^k1 sieve
	size_t indicesCount = (size_t)1 << (TASK_SIZE - 4);        // not all of this will be used

	uint16_t *arraySmall = (uint16_t *)malloc(sizeof(uint16_t) * arraySmallCount);
	uint64_t *arrayLarge = (uint64_t *)malloc(sizeof(uint64_t) * arrayLargeCount);
	uint8_t *arrayIncreases = (uint8_t *)malloc(sizeof(uint8_t) * arrayIncreasesCount);
	uint64_t *indices = (uint64_t *)malloc(sizeof(uint64_t) *indicesCount);

	if ( arraySmall == NULL || arrayLarge == NULL || arrayIncreases == NULL || indices == NULL ) {
		return -1;
	}

	// will be in GPU RAM
	uint16_t *d_arraySmall;
	uint64_t *d_arrayLarge;
	uint64_t *d_arrayLarge2;
	uint8_t *d_arrayIncreases;
	uint64_t *d_indices;

	ret = hipMalloc(&d_arraySmall, sizeof(uint16_t) * arraySmallCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_arrayLarge, sizeof(uint64_t) * arrayLargeCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_arrayLarge2, sizeof(uint64_t) * arrayLarge2Count);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_indices, sizeof(uint64_t) * indicesCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }

	size_t global_work_size1;
	size_t global_work_size1_2;
	size_t global_work_size2;

	global_work_size1 = (size_t)1 << TASK_UNITS;
	global_work_size1_2 = (size_t)1 << TASK_UNITS;

	// copy constant data to GPU
	hipMemcpyToSymbol(HIP_SYMBOL(sieve8), sieve8_host, sizeof(uint8_t)*16);






  /* open the 2^k1 sieve file */

  FILE* fp;
  size_t file_size;

  fp = fopen(file, "rb");

  // Check file size
  // Bytes in sieve file are 2^(k1 - 7)
  fseek(fp, 0, SEEK_END);
  file_size = ftell(fp);
  if ( file_size != ((size_t)1 << (k1 - 7)) ) {
    printf("  error: wrong sieve file!\n");
    return 0;
  }

  /*
    Seek to necessary part of the file
    Note that ((((uint64_t)1 << k1) - 1) & bStart) equals bStart % ((uint64_t)1 << k1)
  */
  __uint128_t bStart = ( (__uint128_t)TASK_ID << TASK_SIZE );
  fseek(fp, ((((uint64_t)1 << k1) - 1) & bStart) >> 7, SEEK_SET);







	/*
	 fill arraySmall[] with part of 2^k1 array, and initialize and arrayLarge[] and arrayIncreases[] to 0,
	 then send to GPU
	*/

	fread(arraySmall, sizeof(uint16_t), arraySmallCount - 1, fp);
	arraySmall[arraySmallCount - 1] = 0xffff;    // to stop kern1 from reading too far
	for (size_t i = 0; i < arrayLargeCount; i++) arrayLarge[i] = 0;
	for (size_t i = 0; i < arrayIncreasesCount; i++) arrayIncreases[i] = 0;

	ret = hipMemcpy(d_arraySmall, arraySmall, sizeof(uint16_t) * arraySmallCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMemcpy(d_arrayLarge, arrayLarge, sizeof(uint64_t) * arrayLargeCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMemcpy(d_arrayIncreases, arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }









	gettimeofday(&tv2, NULL);
	printf("  kernel1 is starting: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

        /* start kernel1 */
	kernel1<<<global_work_size1/(32*multipleOf32), 32*multipleOf32>>>(d_arraySmall, d_arrayLarge, d_arrayIncreases, TASK_ID);










	/* wait for kernel1 to finish (not exactly necessary) */
	hipDeviceSynchronize();

	gettimeofday(&tv2, NULL);
	printf("  kernel1 is finished: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);








	gettimeofday(&tv2, NULL);
	printf("  kernel1_2 is starting: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

        /* start kernel1_2 */
	kernel1_2<<<global_work_size1_2/(32*multipleOf32), 32*multipleOf32>>>(d_arrayLarge2);





	// read arrayIncreases from GPU
	ret = hipMemcpy(arrayIncreases, d_arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount, hipMemcpyDeviceToHost);
	if ( ret != hipSuccess ){ printf("hipMemcpy read error: %s\n", hipGetErrorString(ret)); return -1; }







  /* fill indices[] */

  global_work_size2 = 0;  // for seeing how much work kernel2 has to do

  for (size_t index = 0; index < arrayIncreasesCount; index++) {

        if (arrayIncreases[index]) {
          indices[global_work_size2] = index;
          global_work_size2++;
        }

  }

  printf("Numbers in sieve segment that need testing = %zu\n", global_work_size2);

  /* pad indices[] to make global_work_size2 a multiple of (32 * multipleOf32) */
  for (int j = 0; j < (global_work_size2 % (32 * multipleOf32)); j++) {
    indices[global_work_size2] = (~(uint64_t)0);        // I believe this requires TASK_SIZE < 64 + 4
    global_work_size2++;
  }






	/* write indices[] to GPU */
	ret = hipMemcpy(d_indices, indices, sizeof(uint64_t) * global_work_size2, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }







	/* wait for kernel1_2 to finish */
	hipDeviceSynchronize();

	gettimeofday(&tv2, NULL);
	printf("  kernel1_2 is finished: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);








	/* run kernel 2 */

	gettimeofday(&tv2, NULL);
	printf("  kernel2 is starting: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

	hipEvent_t event;
	hipEventCreateWithFlags(&event, hipEventDisableTiming);
	kernel2<<<global_work_size2/(32*multipleOf32), 32*multipleOf32>>>(d_indices, d_arrayLarge, d_arrayIncreases, d_arrayLarge2, TASK_ID, TASK_ID_KERNEL2);
	hipEventRecord(event);



	/* wait for kernel2 to finish without busy waiting */
	//hipDeviceSynchronize();
	while(hipEventQuery(event) != hipSuccess) {
		usleep(10000);  // sleep for 1/100 of a second
	}




  hipFree(d_arraySmall);
  hipFree(d_arrayLarge);
  hipFree(d_arrayLarge2);
  hipFree(d_arrayIncreases);
  hipFree(d_indices);

  free(arraySmall);
  free(arrayLarge);
  free(arrayIncreases);
  free(indices);

  gettimeofday(&tv2, NULL);
  printf("  Elapsed wall time is %e seconds\n\n",
      (double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));

  return 0;
}
