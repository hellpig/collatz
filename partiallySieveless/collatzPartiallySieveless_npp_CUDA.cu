#include "hip/hip_runtime.h"
/* ******************************************

Sieves of size 2^k are used, where k can be very large!
Not a huge amount of RAM is used.

On Linux, compile via something like...
  nvcc collatzPartiallySieveless_npp_CUDA.cu

You'll need a 64-bit CPU and GPU.

Note that I use the "long long" function strtoull when reading in the arguments.

Currently loads in a sieve file, which must must match the k1 value set in this code.

Currently requires my cuda_uint128.h

k < 81 must be true.





Parts of this file are modified from...
  https://github.com/xbarin02/collatz/blob/master/src/gpuworker/kernel32-precalc.cl



This code requires two arguments...
  ./a.out  TASK_ID_KERNEL2  TASK_ID

Starting at 0, increase TASK_ID by 1 each run until its max
  ( 2^(k - TASK_SIZE) - 1 )
Then, reset TASK_ID and increase TASK_ID_KERNEL2 by 1 (starting at 0).
Don't skip!

Here is a neat way to run this code...
  seq -f %1.0f 0 1048575 | xargs -L 1 -P 1 ./a.out 0 |tee -a log.txt &
To pick up where you left off, change the start (and stop) value of seq.
k, TASK_SIZE, and TASK_SIZE_KERNEL2 should not change between runs.

For each TASK_ID_KERNEL2, 9 * 2 ^ TASK_SIZE_KERNEL2 numbers will be tested,
  but only after each TASK_ID is run from 0 to ( 2^(k - TASK_SIZE) - 1 )
Why the 9? I thought it might help my GPU code, but it only does EXTREMELY SLIGHTLY.
Feel free to get rid of the 9 when h_minimum and h_supremum are defined in kernel2.
You'll also want to get rid of the division by 9 when this host program
  checks if TASK_ID_KERNEL2 will cause certain overflow.
If TASK_ID_KERNEL2 > 0, you'll also want to fix the line in kernel2...
  aMod = 0




This code is not compatible with a watchdog timer.
I recommend that you play around with your device to see if it has a GPU watchdog timer.
It has one if the kernel always stops after a certain amount of time (like 5 seconds).

I would expect that, if a bunch of OVERFLOW messages are printed by kernel2
  within a short period of time, some might be lost due to buffer sizes ??

55247846101001863167 will overflow 128 bits.
This number is over 64-bits, so it cannot be made into an integer literal.
To test 128-bit overflow and using printf() in kernel2,
calculate the following using Python 3, then put it as your TASK_ID_KERNEL2...
  TASK_ID_KERNEL2 = 55247846101001863167 // (9 << TASK_SIZE_KERNEL2)
Then set the following as your TASK_ID...
  remainder = (55247846101001863167 % (9 << TASK_SIZE_KERNEL2))
  TASK_ID = (remainder % (1 << k)) // (1 << TASK_SIZE)
where I am using a double slash, //, for integer division.



(c) 2021 Bradley Knockel

****************************************** */


#include "cuda_uint128.h"

#include <unistd.h>   // for usleep()

#include <sys/time.h>
struct timeval tv1, tv2;





/*
  k < 81
*/

#define SIEVE_LOGSIZE 51
const int k = SIEVE_LOGSIZE;



/*
  The following are in log2...
  For kernel1, which starts making the sieve...
    TASK_UNITS + 8 <= TASK_SIZE <= k
  Will use more than 2^TASK_SIZE bytes of RAM
  Due to the limits of strtoull(), k - TASK_SIZE < 64
  2^TASK_UNITS should be much larger than the number of CUDA Cores you have
  (fraction of 2^k sieve not excluded) * 2^TASK_SIZE should be much larger than the number of CUDA Cores you have
  My CPU-only code prefers a smaller TASK_SIZE so that the task finishes in a reasonable time
*/

#define TASK_SIZE 24
#define TASK_UNITS 16



/*
  The following is in log2...
  TASK_SIZE_KERNEL2 - k should ideally be at least 10
  For kernel2, which tests numbers given a sieve...
  9 * 2^(TASK_SIZE_KERNEL2 + TASK_SIZE - k) numbers will be run by each process
  9 * 2^TASK_SIZE_KERNEL2 numbers will be run total by all processes
*/

#define TASK_SIZE_KERNEL2 67






// For the 2^k1 sieve that speeds up the calculation of the 2^k sieve...
//   TASK_SIZE <= k1 <= k
// The only con of a larger k1 is that the sieve file is harder to create and store,
//   but, once you have the sieve file, use it!
// Especially good values are 32, 35, 37, 40, ...

const int k1 = 37;

const char file[10] = "sieve37";







// integer that is 2^(non-negative integer), so 1, 2, 4, 8, ...
// For setting the number of CUDA threads per block
// Currently, this controls all three kernels
// Note that 2^TASK_UNITS must be at least (multipleOf32 * 2^5), but this shouldn't be a problem
// Note that (2^TASK_UNITS / (multipleOf32 * 2^5) ) cannot exceed the max unsigned int
//   because blockIdx.x is an unsigned int (again, not a problem)

#define multipleOf32 1









/*
  just for kernels
*/

#define LUT_SIZE32 21

#define UINT128_MAX (~(uint128_t)0)

__device__ uint32_t pow3(size_t n)   // returns 3^n
{
	uint32_t r = 1;
	uint32_t b = 3;

	while (n) {
		if (n & 1) {
			r *= b;
		}
		b *= b;
		n >>= 1;
	}

	return r;
}

/* 2^8 sieve */
__constant__ uint8_t sieve8[16];
const uint8_t sieve8_host[16] = {27, 31, 47, 71, 91, 103, 111, 127, 155, 159, 167, 191, 231, 239, 251, 255};










/*
  kernel1 makes 2^k sieve
*/


__global__ void kernel1(
	uint16_t *arraySmall,    // array length is 2^(TASK_SIZE - 8)
	uint64_t *arrayLarge,    // array length is 2^(TASK_SIZE - 3)
	uint8_t *arrayIncreases, // array length is 2^(TASK_SIZE - 4)
	uint64_t TASK_ID
)
{
	size_t id = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;


	__shared__ uint32_t lut[LUT_SIZE32];

	if (threadIdx.x == 0) {
		for (size_t i = 0; i < LUT_SIZE32; ++i) {
			lut[i] = pow3(i);
		}
	}

	__syncthreads();


	size_t i_minimum  = ((size_t)(id + 0) << (TASK_SIZE - TASK_UNITS - 4));
	size_t i_supremum = ((size_t)(id + 1) << (TASK_SIZE - TASK_UNITS - 4));


	/* loop over bits in arraySmall[] */
	size_t pattern = i_minimum >> 4;  /* a group of 256 numbers */
	int bit = 0;                      /* which of the 16 possible numbers in the 2^8 sieve */
	for (size_t index = i_minimum; index < i_supremum; ++index) {

		/* search 2^k1 sieve for next number that could require testing */
		while ( !( arraySmall[pattern] & ((int)1 << bit) ) ) {
			index++;
			if (bit < 15)
				bit++;
			else {
				bit = 0;
				pattern++;
			}
		}
		if (index >= i_supremum) {
			break;
		}

		uint128_t L0 = ((uint128_t)TASK_ID << TASK_SIZE) + (pattern * 256 + sieve8[bit]);

		uint128_t L = L0;
		uint32_t Salpha = 0;    /* sum of alpha, which are the number of increases */

		uint128_t nextL = L0 - 1;
		uint32_t nextSalpha = 0;

		int R = SIEVE_LOGSIZE;  /* counter */
		do {
			/* note that L starts as odd, so we start by increasing */
			++L;
			do {
				//uint32_t alpha = (uint32_t)ctz((uint32_t)L.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = (__ffs((uint32_t)L.lo) - 1);

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				alpha = min(alpha, (uint32_t)R);
				R -= alpha;
				L >>= alpha;
				L *= lut[alpha];
				Salpha += alpha;
				if (R == 0) {
					--L;
					goto next1;
				}
			} while (!(L.lo & 1));
			L.lo--;    // L is odd
			do {
				//size_t beta = ctz((uint32_t)L.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t beta = __ffs((uint32_t)L.lo) - 1;

				beta = min(beta, (uint32_t)R);
				R -= beta;
				L >>= beta;
				if (L < L0) goto next3;
				if (R == 0) goto next1;
			} while (!(L.lo & 1));
		} while (1);

next1:

		/* 
		  Try something else.
		  See if L0 - 1 joins paths in SIEVE_LOGSIZE steps
		  Is it worth doing this ???
		*/

		R = SIEVE_LOGSIZE;
		do {
			/* note that nextL starts as even, so we start by decreasing */
			do {
				//size_t beta = ctz((uint32_t)nextL.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t beta = __ffs((uint32_t)nextL.lo) - 1;

				beta = min(beta, (uint32_t)R);
				R -= beta;
				nextL >>= beta;
				if (R == 0) goto next2;
			} while (!(nextL.lo & 1));
			++nextL;
			do {
				//uint32_t alpha = (size_t)ctz((uint32_t)nextL.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = __ffs((uint32_t)nextL.lo) - 1;

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				alpha = min(alpha, (uint32_t)R);
				R -= alpha;
				nextL >>= alpha;
				nextL *= lut[alpha];
				nextSalpha += alpha;
				if (R == 0) {
					--nextL;
					goto next2;
				}
			} while (!(nextL.lo & 1));
			nextL.lo--;    // nextL is odd
		} while (1);


next2:

		/* only write to RAM if number needs to be tested */
		if ( L == nextL && Salpha == nextSalpha) goto next3;

		arrayLarge[2*index] = L.hi;
		arrayLarge[2*index + 1] = L.lo;
		arrayIncreases[index] = (uint8_t)Salpha;

		//printf(" %llu\n", L0.lo);

next3:

		if (bit < 15)
			bit++;
		else {
			bit = 0;
			pattern++;
		}

	}

}







/*
  kernel2 uses the sieves
*/

__global__ void kernel2(
	uint64_t *indices,         // has a much shorter length than the rest of the arrays
	uint64_t *arrayLarge,      // actually 128-bit integers
	uint8_t *arrayIncreases,
	uint64_t TASK_ID,
	uint64_t TASK_ID_KERNEL2

/*
  index = indices[id] is the only time indices[] will be used
  Only arrayLarge[index*2] and arrayLarge[index*2 + 1] will be used
  Only arrayIncreases[index] will be used
*/

)
{
	size_t id = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;

	__shared__ uint32_t lut[LUT_SIZE32];
	__shared__ uint128_t maxNs[LUT_SIZE32];

	if (threadIdx.x == 0) {
		for (size_t i = 0; i < LUT_SIZE32; ++i) {
			lut[i] = pow3(i);
			//maxNs[i] = UINT128_MAX / lut[i];
		}
		maxNs[0].hi = 18446744073709551615;
		maxNs[0].lo = 18446744073709551615;
		maxNs[1].hi = 6148914691236517205;
		maxNs[1].lo = 6148914691236517205;
		maxNs[2].hi = 2049638230412172401;
		maxNs[2].lo = 14347467612885206812;
		maxNs[3].hi = 683212743470724133;
		maxNs[3].lo = 17080318586768103348;
		maxNs[4].hi = 227737581156908044;
		maxNs[4].lo = 11842354220159218321;
		maxNs[5].hi = 75912527052302681;
		maxNs[5].lo = 10096366097956256645;
		maxNs[6].hi = 25304175684100893;
		maxNs[6].lo = 15663284748458453292;
		maxNs[7].hi = 8434725228033631;
		maxNs[7].lo = 5221094916152817764;
		maxNs[8].hi = 2811575076011210;
		maxNs[8].lo = 7889279663287456460;
		maxNs[9].hi = 937191692003736;
		maxNs[9].lo = 14927589270235519897;
		maxNs[10].hi = 312397230667912;
		maxNs[10].lo = 4975863090078506632;
		maxNs[11].hi = 104132410222637;
		maxNs[11].lo = 7807535721262686082;
		maxNs[12].hi = 34710803407545;
		maxNs[12].lo = 14900341289560596438;
		maxNs[13].hi = 11570267802515;
		maxNs[13].lo = 4966780429853532146;
		maxNs[14].hi = 3856755934171;
		maxNs[14].lo = 13953422859090878459;
		maxNs[15].hi = 1285585311390;
		maxNs[15].lo = 10800055644266810025;
		maxNs[16].hi = 428528437130;
		maxNs[16].lo = 3600018548088936675;
		maxNs[17].hi = 142842812376;
		maxNs[17].lo = 13497835565169346635;
		maxNs[18].hi = 47614270792;
		maxNs[18].lo = 4499278521723115545;
		maxNs[19].hi = 15871423597;
		maxNs[19].lo = 7648674198477555720;
		maxNs[20].hi = 5290474532;
		maxNs[20].lo = 8698472757395702445;
	}

	__syncthreads();

	uint128_t h_minimum = ((uint128_t)(TASK_ID_KERNEL2 + 0)*9 << (TASK_SIZE_KERNEL2 - SIEVE_LOGSIZE));
	uint128_t h_supremum = ((uint128_t)(TASK_ID_KERNEL2 + 1)*9 << (TASK_SIZE_KERNEL2 - SIEVE_LOGSIZE));

	//int cMod = ((uint128_t)1 << SIEVE_LOGSIZE) % 3;
	int cMod = (SIEVE_LOGSIZE & 1) ? 2 : 1 ;



	size_t index = indices[id];
	if (index == (~(uint64_t)0) ) return;

	// deal with the 2^8 sieve that is used to compress the 2^k1 sieve
        //        L0 = TASK_ID * 2^TASK_SIZE             + (index / 16) * 256   + sieve8[index % 16]
	uint128_t L0 = ((uint128_t)TASK_ID << TASK_SIZE) + (((index >> 4) << 8) + sieve8[index & 0xf]);

	int aMod = 0;
	//int bMod = L0 % 3;

	// trick for mod 3 if L0 is uint128_t
	uint64_t r = 0;
	r += (uint32_t)(L0.lo);
	r += (L0.lo >> 32);
	r += (uint32_t)(L0.hi);
	r += (L0.hi >> 32);
        int bMod = r%3;


	/* precalculate */
	uint128_t L;
	L.hi = arrayLarge[2*index];
        L.lo = arrayLarge[2*index + 1];
	uint32_t Salpha = (uint32_t)arrayIncreases[index];


	/* iterate over highest bits */
	for (uint128_t h = h_minimum; h < h_supremum; ++h) {
		uint128_t N0 = (h << SIEVE_LOGSIZE) + L0;
		int N0Mod = (aMod * cMod + bMod) % 3;
		aMod++;
		if (aMod > 2) aMod -= 3;

		while (N0Mod == 2) {
			++h;
			N0 = (h << SIEVE_LOGSIZE) + L0;
			N0Mod = (aMod * cMod + bMod) % 3;
			aMod++;
			if (aMod > 2) aMod -= 3;
		}
		if (h >= h_supremum) {
			break;
		}



		uint128_t N = h;
		uint32_t Salpha0 = Salpha;
		do {
			uint32_t alpha = min(Salpha0, (uint32_t)LUT_SIZE32 - 1);
			N *= lut[alpha];
			Salpha0 -= alpha;
		} while (Salpha0 > 0);
		N += L;


		do {

			/* a "do while" loop won't work because "N >>= 0" isn't defined */
			while (!(N.lo & 1)) {
				//N >>= ctz((uint32_t)N.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 very large */
				uint32_t beta = __ffs((uint32_t)N.lo) - 1;
				if (beta > 32) beta=32;
				N >>= beta;
				
			}
			if (N < N0) {
				goto next;
			}

			++N;
			do {
				//uint32_t alpha = (size_t)ctz((uint32_t)N.lo);
				/* note that __ffs(0) returns 0, and 0 - 1 is caught by min() later */
				uint32_t alpha = (__ffs((uint32_t)N.lo) - 1);

				alpha = min(alpha, (uint32_t)LUT_SIZE32 - 1);
				N >>= alpha;
				if (N > maxNs[alpha]) {
					printf("  OVERFLOW: (%llu << 64) | %llu\n", N0.hi, N0.lo);
					goto next;
				}
				N *= lut[alpha];
			} while (!(N.lo & 1));
			N.lo--;   // N is odd

		} while (1);
next:
		;
	}


}









int main(int argc, char *argv[]) {


  if( argc < 3 ) {
    printf("Too few arguments. Aborting.\n");
    return 0;
  }

  uint64_t TASK_ID_KERNEL2 = (uint64_t)strtoull(argv[1], NULL, 10);

  uint64_t TASK_ID         = (uint64_t)strtoull(argv[2], NULL, 10);

  uint64_t maxTaskID = ((uint64_t)1 << (k - TASK_SIZE));
  if ( TASK_ID >= maxTaskID ) {
    printf("Aborting. TASK_ID must be less than %" PRIu64 "\n", maxTaskID);
    return 0;
  }


  /* Check for 100%-certain overflow.
     This check prevents having to check when doing any pre-calculation when interlacing
     Note that after k steps, for A * 2^k + B...
       B = 2^k - 1 will become 3^k - 1
       and A*2^k will become A*3^k
  */
  __uint128_t temp = 1;  // will equal 3^k
  __uint128_t UINT128_MAX2 = (~(__uint128_t)0);
  for (int i=0; i < k; i++) { temp *= 3; }
  if ( (__uint128_t)TASK_ID_KERNEL2 > ((UINT128_MAX2 - temp) >> (TASK_SIZE_KERNEL2 - k)) / 9 / temp - 1 ) {
    printf("  Well, aren't you ambitious!\n");
    return 0;
  }



  printf("TASK_ID_KERNEL2 = %" PRIu64 "\n", TASK_ID_KERNEL2);
  printf("TASK_ID = %" PRIu64 "\n", TASK_ID);
  printf("TASK_ID must be less than %" PRIu64 "\n", maxTaskID);
  printf("TASK_SIZE_KERNEL2 = %i\n", TASK_SIZE_KERNEL2);
  printf("TASK_SIZE = %i\n", TASK_SIZE);
  printf("  k = %i\n", k);
  printf("  k1 = %i\n", k1);
  printf("\n");
  fflush(stdout);





  // start timing
  gettimeofday(&tv1, NULL);



	/* setup kernel stuff */

	hipError_t ret;

	size_t arraySmallCount = ((size_t)1 << (TASK_SIZE - 8)) + 1;   // each element is 2^8 numbers; add 1 to prevent kern1 from reading too far
	size_t arrayLargeCount = (size_t)1 << (TASK_SIZE - 3);     // two of these are needed per uint128
	size_t arrayIncreasesCount = (size_t)1 << (TASK_SIZE - 4); // one for each bit in portion of 2^k1 sieve
	size_t indicesCount = (size_t)1 << (TASK_SIZE - 4);        // not all of this will be used

	uint16_t *arraySmall = (uint16_t *)malloc(sizeof(uint16_t) * arraySmallCount);
	uint64_t *arrayLarge = (uint64_t *)malloc(sizeof(uint64_t) * arrayLargeCount);
	uint8_t *arrayIncreases = (uint8_t *)malloc(sizeof(uint8_t) * arrayIncreasesCount);
	uint64_t *indices = (uint64_t *)malloc(sizeof(uint64_t) *indicesCount);

	if ( arraySmall == NULL || arrayLarge == NULL || arrayIncreases == NULL || indices == NULL ) {
		return -1;
	}

	// will be in GPU RAM
	uint16_t *d_arraySmall;
	uint64_t *d_arrayLarge;
	uint8_t *d_arrayIncreases;
	uint64_t *d_indices;

	ret = hipMalloc(&d_arraySmall, sizeof(uint16_t) * arraySmallCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_arrayLarge, sizeof(uint64_t) * arrayLargeCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMalloc(&d_indices, sizeof(uint64_t) * indicesCount);
	if ( ret != hipSuccess ){ printf("hipMalloc Error: %s\n", hipGetErrorString(ret)); return -1; }

	size_t global_work_size1;
	size_t global_work_size2;

	global_work_size1 = (size_t)1 << TASK_UNITS;

	// copy constant data to GPU
	hipMemcpyToSymbol(HIP_SYMBOL(sieve8), sieve8_host, sizeof(uint8_t)*16);





  /* open the 2^k1 sieve file */

  FILE* fp;
  size_t file_size;

  fp = fopen(file, "rb");

  // Check file size
  // Bytes in sieve file are 2^(k1 - 7)
  fseek(fp, 0, SEEK_END);
  file_size = ftell(fp);
  if ( file_size != ((size_t)1 << (k1 - 7)) ) {
    printf("  error: wrong sieve file!\n");
    return 0;
  }

  /*
    Seek to necessary part of the file
    Note that ((((uint64_t)1 << k1) - 1) & bStart) equals bStart % ((uint64_t)1 << k1)
  */
  __uint128_t bStart = ( (__uint128_t)TASK_ID << TASK_SIZE );
  fseek(fp, ((((uint64_t)1 << k1) - 1) & bStart) >> 7, SEEK_SET);







	/*
	 fill arraySmall[] with part of 2^k1 array, and initialize and arrayLarge[] and arrayIncreases[] to 0,
	 then send to GPU
	*/

	fread(arraySmall, sizeof(uint16_t), arraySmallCount - 1, fp);
	arraySmall[arraySmallCount - 1] = 0xffff;    // to stop kern1 from reading too far
	for (size_t i = 0; i < arrayLargeCount; i++) arrayLarge[i] = 0;
	for (size_t i = 0; i < arrayIncreasesCount; i++) arrayIncreases[i] = 0;

	ret = hipMemcpy(d_arraySmall, arraySmall, sizeof(uint16_t) * arraySmallCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMemcpy(d_arrayLarge, arrayLarge, sizeof(uint64_t) * arrayLargeCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }
	ret = hipMemcpy(d_arrayIncreases, arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }








	gettimeofday(&tv2, NULL);
	printf("  kernel1 is starting: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

        /* start kernel1 */
	kernel1<<<global_work_size1/(32*multipleOf32), 32*multipleOf32>>>(d_arraySmall, d_arrayLarge, d_arrayIncreases, TASK_ID);










	/* wait for kernel1 to finish */
	hipDeviceSynchronize();

	gettimeofday(&tv2, NULL);
	printf("  kernel1 is finished: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

	// read arrayIncreases from GPU
	ret = hipMemcpy(arrayIncreases, d_arrayIncreases, sizeof(uint8_t) * arrayIncreasesCount, hipMemcpyDeviceToHost);
	if ( ret != hipSuccess ){ printf("hipMemcpy read error: %s\n", hipGetErrorString(ret)); return -1; }






  /* fill indices[] */

  global_work_size2 = 0;  // for seeing how much work kernel2 has to do

  for (size_t index = 0; index < arrayIncreasesCount; index++) {

        if (arrayIncreases[index]) {
          indices[global_work_size2] = index;
          global_work_size2++;
        }

  }

  printf("Numbers in sieve segment that need testing = %zu\n", global_work_size2);

  /* pad indices[] to make global_work_size2 a multiple of (32 * multipleOf32) */
  for (int j = 0; j < (global_work_size2 % (32 * multipleOf32)); j++) {
    indices[global_work_size2] = (~(uint64_t)0);        // I believe this requires TASK_SIZE < 64 + 4
    global_work_size2++;
  }











	/* run kernel 2 */

	ret = hipMemcpy(d_indices, indices, sizeof(uint64_t) * global_work_size2, hipMemcpyHostToDevice);
	if ( ret != hipSuccess ){ printf("hipMemcpy Error: %s\n", hipGetErrorString(ret)); return -1; }

	gettimeofday(&tv2, NULL);
	printf("  kernel2 is starting: %e seconds elapsed\n",
		(double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));
	fflush(stdout);

	hipEvent_t event;
	hipEventCreateWithFlags(&event, hipEventDisableTiming);
	kernel2<<<global_work_size2/(32*multipleOf32), 32*multipleOf32>>>(d_indices, d_arrayLarge, d_arrayIncreases, TASK_ID, TASK_ID_KERNEL2);
	hipEventRecord(event);


	/* wait for kernel2 to finish without busy waiting */
	//hipDeviceSynchronize();
	while(hipEventQuery(event) != hipSuccess) {
		usleep(10000);  // sleep for 1/100 of a second
	}








  gettimeofday(&tv2, NULL);
  printf("  Elapsed wall time is %e seconds\n\n",
      (double)(tv2.tv_usec - tv1.tv_usec) / 1000000.0 + (double)(tv2.tv_sec - tv1.tv_sec));

  hipFree(d_arraySmall);
  hipFree(d_arrayLarge);
  hipFree(d_arrayIncreases);
  hipFree(d_indices);

  free(arraySmall);
  free(arrayLarge);
  free(arrayIncreases);
  free(indices);
  return 0;
}
